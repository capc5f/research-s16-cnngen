#include <stdio.h>
#include <omp.h>
#include <stdlib.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

#include <iostream>
#include <string>
#include <vector>
#include "..//ece8270project/ImageBatch.h"
#include "../layers/InputLayer.h"
#include "../layers/ConvolutionLayer.h"
#include "../layers/PoolingLayer.h"
#include "../layers/ReLULayer.h"
#include "../layers/LRNLayer.h"
#include "../layers/InnerProductLayer.h"
#include "../layers/SoftmaxLayer.h"
#include "../prototxt/Prototxt.h"
#include "../utils/UserInput.h"
#include "../utils/GenerateLayers.h"
#include "../layers/LayerBase.h"
#include "../utils/Utilities.h"

//#include "../layers/layer_helpers/LayerEnums.h"
//#include "../layers/layer_helpers/LayerTypes.h"
//#include "../layers/layer_helpers/Param.h"

#define ITERS 10
#define OMP_NUM_THREADS 16
#define BATCH_SIZE LayerBase::getBatchSize()

hipdnnHandle_t cudnnHandle;			//global handler for cudnn library API
// BATCHSIZE = LayerBase::getBatchSize() 

//\/\/\/\/\/\/ function checks the status of the cuDNN API call and returns error string //\\//\\/\/\/\/ 
void cudnnErrorCheck(hipdnnStatus_t status){
    const char * x;
     if (status!=HIPDNN_STATUS_SUCCESS){
        x = hipdnnGetErrorString(status);
        printf("%s\n",x);
   }
}

//\/\/\/\/\/ function initializes a tensor, filter, conv layer with random floating point values between 0 and 1 /\/\/\/\/\/\/
void initializeMatrix(float * entity,int size){
    int i = 0;
    #pragma omp parallel for    
       for (i=0;i<size;i++){
            entity[i]= (float)((float) rand() / (float)RAND_MAX);
       }

}

//\/\/\/\/// function prints if allocation fails ///\/\/\/\///
void allocCheck(float * x){
    if (x==NULL)
        puts("memory allocation failure");
}

//\/\/\/\/// makes sure cudamemcpy works ///\/\/\/\///
void cudaErrCheck(hipError_t err){
    if (err!= hipSuccess){
        printf("%s\n",hipGetErrorString(err));
    }
}


///\/\/\/\/\/\\/\/\/\/ Tensor Superclass ///\/\/\/\/\/\/\/\/// 
class Tensor {
	public: 
		~Tensor();
		hipdnnTensorDescriptor_t tensorDesc;
		hipdnnTensorFormat_t tensFormat;
		hipdnnDataType_t dataType;
		float *h_tensor,*d_tensor;
		size_t size;
		int height, width, depth, n, c, h, w;
		LayerType type;
		Tensor(int, int ,int, int);
		Tensor();
		virtual void doAction(hipdnnTensorDescriptor_t tensorDesc2, float *d_tensor2){};
};

										///\/\/\/\/ Default Constructor ///\/\/\/\/
Tensor::Tensor(){
	d_tensor = NULL;
	h_tensor = NULL;
}

										///\/\/\/\/ Constructor ///\/\/\/\/
Tensor::Tensor(int n,int c,int h,int w){
	printf("Tensor vals: n = %d c = %d h = %d w = %d\t", n, c ,h, w);
	d_tensor = NULL;
	h_tensor = NULL;
	this->n = n;
	this->c = c;
	this->h = h;
	this->w = w;
	size = n*c*h*w;  

	h_tensor = (float *)malloc(sizeof(float)*size);			
	hipMalloc((void **)&(d_tensor),sizeof(float)*(size));
	allocCheck(h_tensor);
	allocCheck(d_tensor);

    initializeMatrix(h_tensor,size);				
	cudaErrCheck(hipMemcpy(d_tensor,h_tensor,(sizeof(float)*size),hipMemcpyHostToDevice)); //copy initial weights

    cudnnErrorCheck(hipdnnCreateTensorDescriptor(&tensorDesc));
    tensFormat = HIPDNN_TENSOR_NCHW;
    dataType = HIPDNN_DATA_FLOAT;
    cudnnErrorCheck(hipdnnSetTensor4dDescriptor(tensorDesc, tensFormat, dataType, n, c, h, w));
	puts("Tensor Created");
}

										///\/\/\/\/\/ Destructor ///\/\/\/\/\/\/\/\//
Tensor::~Tensor(){
	free(h_tensor);
	hipFree(d_tensor);
	cudnnErrorCheck(hipdnnDestroyTensorDescriptor(tensorDesc));
}

///\/\/\/\/\/\/\/\/\/ Filter Class ///\/\/\/\/\/\///
class Filter{
	public:
		~Filter();
    	int fn,fc,fh,fw;
		size_t filterSize;
		hipdnnDataType_t dType;
    	float *h_filter, *d_filter;	
		hipdnnFilterDescriptor_t filterDesc;
    	Filter(ConvolutionLayer *);
		void doAction(hipdnnTensorDescriptor_t, float *){};
};

										///\/\/\/\/\/\/ Filter Constructor ///\/\/\/\/\/\/
Filter::Filter(ConvolutionLayer* convolutionLayer) {
	fn = BATCH_SIZE;
	fc = convolutionLayer->getFilterDepth();
	fh = convolutionLayer->getFilterSize();
	fw = convolutionLayer->getFilterSize();	
	filterSize = fn*fc*fh*fw;
	dType = HIPDNN_DATA_FLOAT;

	h_filter = (float *)malloc(sizeof(float)*filterSize);
    hipMalloc((void **)&(d_filter),sizeof(float)*(filterSize));
    allocCheck(h_filter);
    allocCheck(d_filter);

    initializeMatrix(h_filter,filterSize);                
    cudaErrCheck(hipMemcpy(d_filter,h_filter,(sizeof(float)*filterSize),hipMemcpyHostToDevice)); //copy initial weights	
    cudnnErrorCheck(hipdnnCreateFilterDescriptor(&filterDesc));
    cudnnErrorCheck(hipdnnSetFilter4dDescriptor(filterDesc,dType,fn,fc,fh,fw)); 
}

										///\/\/\/\/\/\/ Destructor ///\/\/\/\/\/
Filter::~Filter(){
	free(h_filter);
	hipFree(d_filter);
	cudnnErrorCheck(hipdnnDestroyFilterDescriptor(filterDesc));
}

///\/\/\/\/\/\/\/\/\/ Convolution Layer Class ///\/\/\/\/\/\///
class ConvLayer: public Tensor{
	public:
		~ConvLayer();
		size_t workspaceSize;
		float * d_space;
		const void *alpha,*beta;
		hipdnnConvolutionDescriptor_t convLayer;
		
		Filter *filter;
		hipdnnConvolutionMode_t mode;
		hipdnnConvolutionFwdAlgo_t convAlgorithm;	

		ConvLayer(LayerBase *);
		void doAction(hipdnnTensorDescriptor_t,float *);
};

										///\/\/\/\/\/\/ Convolution Constructor ///\/\/\/\/\/\/
ConvLayer::ConvLayer(LayerBase *baseLayer) : Tensor(BATCH_SIZE,baseLayer->getInputDepth(),baseLayer->getInputHeight(),baseLayer->getInputWidth()){
	alpha = ((ConvolutionLayer *)baseLayer)->getAlpha();
	beta = ((ConvolutionLayer *)baseLayer)->getBeta();
	workspaceSize = 0;
	d_space = NULL;	
    mode = HIPDNN_CONVOLUTION;
    convAlgorithm = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
	cudnnErrorCheck(hipdnnCreateConvolutionDescriptor(&convLayer));

	filter = new Filter((ConvolutionLayer *)baseLayer);

    cudnnErrorCheck(hipdnnSetConvolution2dDescriptor(convLayer,((ConvolutionLayer *)baseLayer)->getPadding(),((ConvolutionLayer *)baseLayer)->getPadding(),((ConvolutionLayer *)baseLayer)->getStride(),((ConvolutionLayer *)baseLayer)->getStride(),((ConvolutionLayer *)baseLayer)->getUpscaleX(),((ConvolutionLayer *)baseLayer)->getUpscaleY(),mode));
}

										///\/\/\/\/\/\/ Convolution Action ///\/\/\/\/\///
void ConvLayer::doAction(hipdnnTensorDescriptor_t tensorDesc2, float *d_tensor2){
	cudnnErrorCheck(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,tensorDesc,filter->filterDesc,convLayer,tensorDesc2,convAlgorithm,&workspaceSize)); 
    hipMalloc((void**) &d_space, workspaceSize);
    allocCheck(d_space);
	hipdnnConvolutionForward(cudnnHandle,alpha,tensorDesc, d_tensor, filter->filterDesc,filter->d_filter , convLayer, convAlgorithm,d_space,workspaceSize,beta,tensorDesc2,d_tensor2);
	hipFree(d_space);
}

										///\/\/\/\/\/\/ Destructor ///\/\/\/\/\/\/												
ConvLayer::~ConvLayer(){
//	hipFree(d_space);
	cudnnErrorCheck(hipdnnDestroyConvolutionDescriptor(convLayer));
}

///\/\/\/\/\/\/\/\/\/\/ Pooling Layer Class ///\/\/\/\/\/\/\///
class PoolLayer: public Tensor{
	public:
		~PoolLayer();
		hipdnnDataType_t dataType;
		hipdnnPoolingDescriptor_t poolDesc;
		hipdnnPoolingMode_t poolMode;
		int poolHeight,poolWidth,vertPad,horizPad,vertStride,horizStride;
		float a,b;
        const void *alpha,*beta;
		PoolLayer(LayerBase *);
		void doAction(hipdnnTensorDescriptor_t,float *);
};

										///\/\/\/\/ Constructor ///\/\/\/\/
PoolLayer::PoolLayer(LayerBase *baseLayer) : Tensor(BATCH_SIZE,baseLayer->getInputDepth(),baseLayer->getInputHeight(),baseLayer->getInputWidth()){
	
	a=0.0; b=0.0;
	alpha = &a;
	beta = &b;													
	poolWidth = (baseLayer)->getInputWidth();
	poolHeight = (baseLayer)->getInputHeight();
	vertPad = ((PoolingLayer *)baseLayer)->getPadding();
	horizPad = ((PoolingLayer *)baseLayer)->getPadding();
	vertStride = ((PoolingLayer *)baseLayer)->getStride();
	horizStride = ((PoolingLayer *)baseLayer)->getStride();
	poolMode = HIPDNN_POOLING_MAX;
	
	cudnnErrorCheck(hipdnnCreatePoolingDescriptor(&poolDesc)); 
    cudnnErrorCheck(hipdnnSetPooling2dDescriptor(poolDesc, poolMode, poolHeight, poolWidth, vertPad, horizPad, vertStride, horizStride));
	
}

										///\/\/\/\/\/\/ Pooling Action ///\/\/\/\/\/
void PoolLayer::doAction(hipdnnTensorDescriptor_t tensorDesc2, float *d_tensor2){
	cudnnErrorCheck(hipdnnPoolingForward(cudnnHandle,poolDesc,alpha,tensorDesc,d_tensor,beta,tensorDesc2, d_tensor2));
}

										///\/\/\/\/\/\/ Destructor ///\/\/\/\/\/
PoolLayer::~PoolLayer(){
	cudnnErrorCheck(hipdnnDestroyPoolingDescriptor(poolDesc));	
}

///\/\/\/\/\/\/\/\/ Activation Layer Class ///\/\/\/\/\//\/\/\////
class ActivationLayer: public Tensor{
	public:
		~ActivationLayer();
		float a,b;
		const void *alpha, *beta;
		hipdnnActivationMode_t activMode;
		ActivationLayer(LayerBase *);
		void doAction(hipdnnTensorDescriptor_t, float*);	
};
											///\/\/\/\/ Constructor ///\/\/\/\/
ActivationLayer::ActivationLayer(LayerBase *baseLayer) : Tensor(BATCH_SIZE,baseLayer->getInputDepth(),baseLayer->getInputHeight(),baseLayer->getInputWidth()){
	activMode = HIPDNN_ACTIVATION_RELU;
    a=0.0; b=0.0;
    alpha = &a;
    beta = &b;
}

											///\/\/\// Activation Action ///\/\///
void ActivationLayer::doAction(hipdnnTensorDescriptor_t tensorDesc2, float *d_tensor2){
	cudnnErrorCheck(hipdnnActivationForward(cudnnHandle,activMode,alpha,tensorDesc,d_tensor,beta,tensorDesc2,d_tensor2));
//	cudnnErrorCheck(hipdnnActivationForward(cudnnHandle,activMode,alpha,tensorDesc2,d_tensor2,beta,tensorDesc,d_tensor));
}

											///\/\/\/\/ Destructor ///\/\/\/\/\/\/\/
ActivationLayer::~ActivationLayer(){

}

///\/\/\/\/\/\/\/\/ Fully Connected Filter ///\/\/\/\/\/\/\/\/////
class FCFilter{
    public:
        ~FCFilter();
        int fn,fc,fh,fw;
        size_t filterSize;
        hipdnnDataType_t dType;
        float *h_filter, *d_filter; 
        hipdnnFilterDescriptor_t filterDesc;
        FCFilter(InnerProductLayer *);
        void doAction(hipdnnTensorDescriptor_t, float *){};
};

                                        	///\/\/\/\/\/ FCFilter Constructor ///\/\/\/\/\/\/
FCFilter::FCFilter(InnerProductLayer *fullyConnectedLayer){
    fn = BATCH_SIZE;
    fc = fullyConnectedLayer->getInputDepth();
    fh = fullyConnectedLayer->getInputHeight();
    fw = fullyConnectedLayer->getInputWidth();
    filterSize = fn*fc*fh*fw;
    dType = HIPDNN_DATA_FLOAT;

    h_filter = (float *)malloc(sizeof(float)*filterSize);
    hipMalloc((void **)&(d_filter),sizeof(float)*(filterSize));
    allocCheck(h_filter);
    allocCheck(d_filter);

    initializeMatrix(h_filter,filterSize);                
    cudaErrCheck(hipMemcpy(d_filter,h_filter,(sizeof(float)*filterSize),hipMemcpyHostToDevice));    
    cudnnErrorCheck(hipdnnCreateFilterDescriptor(&filterDesc));
    cudnnErrorCheck(hipdnnSetFilter4dDescriptor(filterDesc,dType,fn,fc,fh,fw)); 
}

											///\/\/\/\/\/ Destructor ///\/\/\/\/\////
FCFilter::~FCFilter(){
    free(h_filter);
    hipFree(d_filter);
	cudnnErrorCheck(hipdnnDestroyFilterDescriptor(filterDesc));
}

///\/\/\/\/\/\/\/\/ Fully Connected Layer ///\/\/\/\/\/\/\/\////
class FCLayer: public Tensor{
    public:
        ~FCLayer();
        size_t workspaceSize;
        float * d_space;
        const void *alpha,*beta;
        hipdnnConvolutionDescriptor_t convLayer;     
        FCFilter *filter;
        hipdnnConvolutionMode_t mode;
        hipdnnConvolutionFwdAlgo_t convAlgorithm;
        FCLayer(LayerBase *);
        void doAction(hipdnnTensorDescriptor_t,float *);
};

                                        	///\/\/\/\/\/\/ FC Constructor ///\/\/\/\/\/\/
FCLayer::FCLayer(LayerBase *baseLayer) : Tensor(BATCH_SIZE,baseLayer->getInputDepth(),baseLayer->getInputHeight(),baseLayer->getInputWidth()){
    alpha = ((InnerProductLayer *)baseLayer)->getAlpha();
    beta = ((InnerProductLayer *)baseLayer)->getBeta();
    workspaceSize = 0;
    d_space = NULL;
    mode = HIPDNN_CONVOLUTION;
    convAlgorithm = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
    cudnnErrorCheck(hipdnnCreateConvolutionDescriptor(&convLayer));
    filter = new FCFilter((InnerProductLayer *)baseLayer); 					
	cudnnErrorCheck(hipdnnSetConvolution2dDescriptor(convLayer,0,0,1,1,1,1,mode));
}

                                        	///\/\/\/\/\/\/ FC Action ///\/\/\/\/\///
void FCLayer::doAction(hipdnnTensorDescriptor_t tensorDesc2, float *d_tensor2){
   cudnnErrorCheck(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,tensorDesc,filter->filterDesc,convLayer,tensorDesc2,convAlgorithm,&workspaceSize)); 
   hipMalloc((void**) &d_space, workspaceSize);
   allocCheck(d_space);
   hipdnnConvolutionForward(cudnnHandle,alpha,tensorDesc, d_tensor, filter->filterDesc,filter->d_filter , convLayer, convAlgorithm,d_space,workspaceSize,beta,tensorDesc2,d_tensor2);
	hipFree(d_space);
}

											///\/\/\//\/\/\/ Destructor ///\/\/\/\/\/\////
FCLayer::~FCLayer(){
//    hipFree(d_space);
	cudnnErrorCheck(hipdnnDestroyConvolutionDescriptor(convLayer));
}
//////////////////////////////////////////////   MAIN PROGRAM \\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\//
int main(){
	//define and instantiate number of layers recieved

	cudnnErrorCheck(hipdnnCreate(&cudnnHandle));
	int i;
	UserInput in(false);
	if ( !in.isValidInput() ) {
		std::cout << "Exiting program.." << std::endl;
		return EXIT_FAILURE;
	}	
	GenerateLayers gen(in);
	std::vector <LayerBase *> layers;
	layers = gen.getLayerList();		//layers = alexnet now
	displayPrototxtOutput(in, layers, true);
	LayerType layerType; 

	Tensor * layerArray[layers.size()+1];			//to allow for output tensor
	#pragma omp parallel for						//////////layer and tensor setup, build network 
		for (i=0;i<layers.size();i++){				
			//generate array of cudnn layers
			layerType = (layers.at(i))->getLayerType();
			switch (layerType){
				case INPUT:
						layerArray[i] = new Tensor(BATCH_SIZE,layers.front()->getInputDepth(),layers.front()->getInputHeight(),layers.front()->getInputWidth());
						puts("input skipped");
						break;
				case CONVOLUTION:
						layerArray[i] = new ConvLayer(layers.at(i));
						puts("conv created ");
						break;
				case POOLING:
						
						layerArray[i] = new PoolLayer(layers.at(i));
						puts("pool created ");
						break;
				case RELU:
						layerArray[i] = new ActivationLayer(layers.at(i));
						puts("relu created ");
						break;
				case INNER_PRODUCT:
						layerArray[i] = new FCLayer(layers.at(i));
						puts("fc created ");
						break;
				default:
						puts("error");
						break;
			}

		}
		puts("before layerArray");
		layerArray[i] = new Tensor(BATCH_SIZE,layers.back()->getOutputDepth(),layers.back()->getOutputHeight(),layers.back()->getOutputWidth());
		puts("After layerArray, before ImageBatch vec init");
	//std::vector <ImageBatch *> batchList = buildBatchList(in,BATCH_SIZE);
		puts("After batc vec init");
	int x = 0,y=0;
	for(y=0;y<10;y++){
	std::vector <ImageBatch *> batchList = buildBatchList(in, BATCH_SIZE);
	
	for(x=0;x<batchList.size();x++){		
		printf("Batch number %d \n",x);
		ImageBatch *newBatch = batchList.at(x);
		for (i=0;i<layers.size();i++){
            //generate array of cudnn layers
            layerType = (layers.at(i))->getLayerType();
            switch (layerType){
                case INPUT: 
						//set convolution input descriptors as the input images		
						cudaErrCheck(hipMemcpy(layerArray[i+1]->d_tensor,newBatch->get(),(sizeof(float)*newBatch->length()),hipMemcpyHostToDevice));	
                        //create convolution layer for input image
  
                        puts("input finished");
                        break;
                case CONVOLUTION:
//						puts("b4");
                        layerArray[i]->doAction(layerArray[i+1]->tensorDesc,layerArray[i+1]->d_tensor);
                        puts("conv finished ");
                        break;
                case POOLING:
//						puts("b4");
						layerArray[i]->doAction(layerArray[i+1]->tensorDesc,layerArray[i+1]->d_tensor);                        
                        puts("pool finished ");
                        break;
                case RELU:
//						puts("b4");
                        layerArray[i]->doAction(layerArray[i+1]->tensorDesc,layerArray[i+1]->d_tensor);
                        puts("relu finished ");
                        break;
                case INNER_PRODUCT:
//						puts("b4");
                        layerArray[i]->doAction(layerArray[i+1]->tensorDesc,layerArray[i+1]->d_tensor);
                        puts("fc finished ");
                        break;
                default:
                        puts("error");
                        break;
			}
         }
		newBatch->clean();
	}
	
		destroyBatchList(batchList);
	}
	//destroyBatchList(batchList);
	layers.clear();
	
	return 0;
}
